#include "hip/hip_runtime.h"
/******************************************************************************
* Copyright 2013 PopWi Technology Group, Inc. (PTG)
*
* This file is proprietary and exclusively owned by PTG or its associates.
* This document is protected by international and domestic patents where
* applicable. All rights reserved.
*
******************************************************************************/

//#include <hip/hip_complex.h>
#include <complex>
#include <iostream>
#include <stdexcept>
#include "utils.hpp"

using namespace std;

#define MAX_THREADS_PER_BLOCK 1024


struct hipComplex
{
	float r;
	float i;

	__device__ hipComplex( float a, float b ) : r(a), i(b) {}

	__device__ float magnitude2( void )
	{
		return r * r + i * i;
	}

	__device__ hipComplex operator*(const hipComplex& a)
	{
		return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
	}

	__device__ hipComplex operator+(const hipComplex& a)
	{
		return hipComplex(r+a.r, i+a.i);
	}

	__device__ hipComplex operator+=(const hipComplex& a)
	{
		return hipComplex(r+a.r, i+a.i);
	}
};



__global__ void deconvolve(hipComplex *pn, hipComplex *data, 
	hipComplex *old_data, float *product, int pn_len)
{
	/*int threadsPerBlock = blockDim.x * blockDim.y;
	int blockId = blockIdx.x + (blockIdx.y * gridDim.x);
	int threadId = threadIdx.x + (threadIdx.y * blockDim.x);
	int globalIdx = (blockId * threadsPerBlock) + threadIdx.x + (threadIdx.y * blockDim.x);
	int n;
	int pn_idx;*/

	//int i = blockIdx.x * blockDim.x + threadIdx.x;

	//if(globalIdx >= pn_len) return;

	//hipComplex s = hipComplex(0.0, 0.0);

	//product[i] = (float)i;

	// TODO: this isn't the real deconvolve algo. PN loops back on itself here...
	/*for( n = 0; n < pn_len; n++){
		pn_idx = (globalIdx + n) % pn_len;
		s += data[n] * pn[pn_idx];
	}*/

	//product[globalIdx] = s.magnitude2();
	//product[globalIdx] = 1; // ##### DEBUG OUTPUT - FIXME! ####### 

	/* old deconvolve ref.... 
	int i = threadIdx.x;
	int N = blockDim.x;
	int I = N - i;
	int n;
	hipComplex s = hipComplex(0.0, 0.0);

	for( n = 0; n < I; n++)
		s += data[n] * pn[n + i];
	for( n = i; n < N; n++)
		s += old_data[n] * pn[n + I];

	product[i] = s.magnitude2();
	*/
}

extern "C"
{	
	hipComplex *d_pcode;
	hipComplex *d_data1;
	hipComplex *d_data2;
	float *d_prod1;
	int h_buf_idx;
	size_t h_len;


	void start_deconvolve(complex<float> *h_data, float *h_product)
	{
		hipComplex *d_data_a;
		hipComplex *d_data_b;

		// Double buffer switch
		if( 1 == h_buf_idx )
		{
			d_data_a = d_data1;
			d_data_b = d_data2;
			h_buf_idx = 0;
		}
		else
		{
			d_data_a = d_data2;
			d_data_b = d_data1;
			h_buf_idx = 1;
		}

		for(unsigned n = 0; n < 65535; n++)
			h_product[n] = (float)n;

		// copy new host data into device memory
		hipMemcpy(d_data_a, h_data, h_len, hipMemcpyHostToDevice);

		// Task the SM's
		deconvolve<<<64, 1024>>>(d_pcode, d_data_a, d_data_b, d_prod1, h_len);
  		checkCudaErrors(hipGetLastError());
		
	    // Copy results to host
		//hipMemcpy(h_product, d_prod1, h_len * sizeof(float), hipMemcpyDeviceToHost);
	}


	void init_deconvolve(complex<float> *h_pn, size_t len)
	{
		h_len = len;

		// verify that we're a multiple of samples of a thread block
		if( 0 != (h_len % MAX_THREADS_PER_BLOCK) )
			throw runtime_error("[POPGPU] - sample length needs to be multiple of block size.\r\n");

		// allocate CUDA memory
		checkCudaErrors(hipMalloc(&d_pcode, h_len * sizeof(hipComplex)));
		checkCudaErrors(hipMalloc(&d_data1, h_len * sizeof(hipComplex)));
		checkCudaErrors(hipMalloc(&d_data2, h_len * sizeof(hipComplex)));
		checkCudaErrors(hipMalloc(&d_prod1, h_len * sizeof(float)));

		// initialize CUDA memory
		checkCudaErrors(hipMemcpy(d_pcode, h_pn, h_len, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemset(d_data1, 0, h_len * sizeof(hipComplex)));
		checkCudaErrors(hipMemset(d_data2, 0, h_len * sizeof(hipComplex)));		
		checkCudaErrors(hipMemset(d_prod1, 0, h_len * sizeof(float)));
	}

	//Free all the memory that we allocated
	//TODO: check that this is comprehensive
	void cleanup() {
	  checkCudaErrors(hipFree(d_pcode));
	  checkCudaErrors(hipFree(d_data1));
	  checkCudaErrors(hipFree(d_data2));
	  checkCudaErrors(hipFree(d_prod1));
	}

}