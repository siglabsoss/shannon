#include "hip/hip_runtime.h"
/******************************************************************************
* Copyright 2013 PopWi Technology Group, Inc. (PTG)
*
* This file is proprietary and exclusively owned by PTG or its associates.
* This document is protected by international and domestic patents where
* applicable. All rights reserved.
*
******************************************************************************/

//#include <hip/hip_complex.h>
#include <complex>
#include <iostream>
#include <stdexcept>
#include "utils.hpp"

using namespace std;

#define MAX_THREADS_PER_BLOCK 1024


struct hipComplex
{
	float r;
	float i;

	__device__ hipComplex( float a, float b ) : r(a), i(b) {}

	__device__ float magnitude2( void )
	{
		return r * r + i * i;
	}

	__device__ hipComplex operator*(const hipComplex& a)
	{
		return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
	}

	__device__ hipComplex operator+(const hipComplex& a)
	{
		return hipComplex(r+a.r, i+a.i);
	}

	__device__ hipComplex operator+=(const hipComplex& a)
	{
		return hipComplex(r+a.r, i+a.i);
	}
};



__global__ void deconvolve(hipComplex *pn, hipComplex *data,
	float *product, int pn_len)
{
	int n;
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	hipComplex s = hipComplex(0.0, 0.0);

	for( n = 0; n < pn_len; n++)
		s += data[n + i] * pn[n];

	product[i] = s.magnitude2();
}

extern "C"
{	
	hipComplex *d_prncode;
	hipComplex *d_dataold;
	hipComplex *d_datanew;
	float *d_product;
	size_t h_len; ///< length of data in samples


	void start_deconvolve(complex<float> *h_data, float *h_product)
	{
		// copy new memory to old
		hipMemcpy(d_dataold, d_datanew, h_len * sizeof(hipComplex), hipMemcpyDeviceToDevice);

		// copy new host data into device memory
		hipMemcpy(d_datanew, h_data, h_len * sizeof(hipComplex), hipMemcpyHostToDevice);

		// Task the SM's
		deconvolve<<<64, 1024>>>(d_prncode, d_dataold, d_product, h_len);
  		checkCudaErrors(hipGetLastError());
		
	    // Copy results to host
		hipMemcpy(h_product, d_product, h_len * sizeof(float), hipMemcpyDeviceToHost);
	}


	void init_deconvolve(complex<float> *h_pn, size_t len)
	{
		h_len = len;

		// verify that we're a multiple of samples of a thread block
		if( 0 != (h_len % MAX_THREADS_PER_BLOCK) )
			throw runtime_error("[POPGPU] - sample length needs to be multiple of block size.\r\n");

		// allocate CUDA memory
		checkCudaErrors(hipMalloc(&d_prncode, h_len * sizeof(hipComplex)));
		checkCudaErrors(hipMalloc(&d_dataold, h_len * sizeof(hipComplex) * 2));
		d_datanew = d_dataold + h_len; ///< make this sequential to old data
		checkCudaErrors(hipMalloc(&d_product, h_len * sizeof(float)));

		// initialize CUDA memory
		checkCudaErrors(hipMemcpy(d_prncode, h_pn, h_len, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemset(d_dataold, 0, h_len * sizeof(hipComplex)));
		checkCudaErrors(hipMemset(d_datanew, 0, h_len * sizeof(hipComplex)));		
		checkCudaErrors(hipMemset(d_product, 0, h_len * sizeof(float)));
	}

	//Free all the memory that we allocated
	//TODO: check that this is comprehensive
	void cleanup() {
	  checkCudaErrors(hipFree(d_prncode));
	  checkCudaErrors(hipFree(d_dataold));
	  checkCudaErrors(hipFree(d_datanew));
	  checkCudaErrors(hipFree(d_product));
	}

}