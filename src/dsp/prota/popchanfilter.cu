#include "hip/hip_runtime.h"
/******************************************************************************
* Copyright 2013 PopWi Technology Group, Inc. (PTG)
*
* This file is proprietary and exclusively owned by PTG or its associates.
* This document is protected by international and domestic patents where
* applicable. All rights reserved.
*
******************************************************************************/

// Bad practice, but I can't get multiple cuda files to link http://stackoverflow.com/questions/13683575/cuda-5-0-separate-compilation-of-library-with-cmake
#include <dsp/prota/popdeconvolve.cu>
#include <dsp/common/poptypes.cu>


#include <iostream>
#include <stdexcept>
#include <cstdio>
#include <cmath>
#include "dsp/utils.hpp"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/tuple.h>
#include <thrust/reduce.h>
#include <thrust/fill.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <cstdlib>
#include <time.h>

#include <dsp/common/poptypes.cuh>

#include <dsp/prota/popchanfilter.cuh>

#include "core/basestationfreq.h"



using namespace std;
using namespace thrust;


__global__ void rolling_scalar_multiply(popComplex *in, popComplex *cfc, popComplex *out, int len)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int I = blockDim.x * gridDim.x;
	
	int fsearchbin = (I / len);
	int fidx = (i / len) - (fsearchbin / 2); // frequency modulation index
	int b = i % len; // fft bin
	int cidx = (b + fidx + len) % len;

	out[i] = in[b] * cfc[cidx];
}

__device__ unsigned IFloatFlip(unsigned f)
{
	unsigned mask = ((f >> 31) - 1) | 0x80000000;
	return f ^ mask;
}

__device__ unsigned FloatFlip(unsigned f)
{
	unsigned mask = -signed(f >> 31) | 0x80000000;
	return f ^ mask;
}



extern "C"
{	
	popComplex* d_dataa;
	popComplex* d_datab;
	popComplex* d_datac;
	popComplex* d_datad;
	hipfftHandle plan1;
	hipfftHandle plan2;
	hipfftHandle many_plan;
	size_t g_len_chan; ///< length of time series in samples (CHAN_SIZE)
	size_t g_len_fft; ///< length of fft in samples (FFT_SIZE)
//	size_t g_start_chan;


	size_t gpu_channel_split(const complex<double> *h_data, complex<double> *out)
	{
		//double ch_start, ch_end, ch_ctr;


		// copy new host data into device memory for fft (this is for all data and all channels)
		hipMemcpy(d_dataa, h_data, g_len_fft * sizeof(popComplex), hipMemcpyHostToDevice);
		hipDeviceSynchronize();

		// perform FFT on entire spectrum
		hipfftExecZ2Z(plan1, (hipfftDoubleComplex*)d_dataa, (hipfftDoubleComplex*)d_datab, HIPFFT_FORWARD);
		hipDeviceSynchronize();




/*		ch_start = 903626953 + (3200000 / (double)g_len_fft * (double)g_start_chan) - 1600000;
		ch_end = 903626953 + (3200000 / (double)g_len_fft * ((double)g_start_chan + 1040)) - 1600000;
		ch_ctr = (ch_start + ch_end) / 2.0;*/
		//printf("channel start: %f (%llu), end: %f, ctr: %f\r\n", ch_start, g_start_chan, ch_end, ch_ctr);



		// calculate small bin side-band size (same for every channel)
		unsigned small_bin_sideband = g_len_chan / 2;




		// do 50 cuda mem copies
		for( int c = 0; c < 50; c++ )
		{
			// shift zero-frequency component to center of spectrum ( calculate the bin in which the fft starts adjusting for the fact that the complex fft has 0 freq in the center)
			unsigned small_bin_start = bsf_zero_shift_channel_fbin_low(c); //(g_start_chan + (g_len_fft/2)) % g_len_fft;

			//FIXME: start memory in d_datac at 0

			// chop spectrum up into 50 spreading channels low side-band
			hipMemcpy(d_datac + small_bin_start,
					   d_datab + small_bin_start + small_bin_sideband,

					   small_bin_sideband * sizeof(popComplex),
					   hipMemcpyDeviceToDevice);
			// chop spectrum up into 50 spreading channels high side-band
			hipMemcpy(d_datac + small_bin_start + small_bin_sideband,
					   d_datab + small_bin_start,

					   small_bin_sideband * sizeof(popComplex),
					   hipMemcpyDeviceToDevice);

		}
		
		hipDeviceSynchronize();


		// put back into time domain
		hipfftExecZ2Z(many_plan, (hipfftDoubleComplex*)d_datac, (hipfftDoubleComplex*)d_datad, HIPFFT_BACKWARD);
		checkCudaErrors(hipGetLastError());
		hipDeviceSynchronize();
  		checkCudaErrors(hipGetLastError());

		unsigned channel = 9;

  		unsigned data_range_low = bsf_zero_shift_channel_fbin_low(channel);
  		unsigned data_length = bsf_zero_shift_channel_fbin_low(channel);

  		// Copy results to host
		hipMemcpy(out, d_datad + data_range_low, g_len_chan * sizeof(popComplex), hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		
  		return 0;
	}


	void init_deconvolve(size_t len_fft, size_t len_chan)
	{
		g_len_chan = len_chan;
		g_len_fft = len_fft;

		// allocate CUDA memory
		checkCudaErrors(hipMalloc(&d_dataa, g_len_fft * sizeof(popComplex)));
		checkCudaErrors(hipMalloc(&d_datab, g_len_fft * sizeof(popComplex)));
		checkCudaErrors(hipMalloc(&d_datac, g_len_fft * sizeof(popComplex)));
		checkCudaErrors(hipMalloc(&d_datad, g_len_fft * sizeof(popComplex)));

		// initialize CUDA memory
		checkCudaErrors(hipMemset(d_dataa, 0, g_len_fft * sizeof(popComplex)));
		checkCudaErrors(hipMemset(d_datab, 0, g_len_fft * sizeof(popComplex)));
		checkCudaErrors(hipMemset(d_datac, 0, g_len_fft * sizeof(popComplex)));
		checkCudaErrors(hipMemset(d_datad, 0, g_len_fft * sizeof(popComplex)));

	    // setup FFT plans
	    hipfftPlan1d(&plan1, g_len_fft, HIPFFT_Z2Z, 1);
	    hipfftPlan1d(&plan2, g_len_chan, HIPFFT_Z2Z, 1);

	    // Setup multiple FFT plan
	    int dimension_size[1];
	    dimension_size[0] = g_len_chan; // how big is the first dimension of the transform

//	    http://docs.nvidia.com/cuda/cufft/#function-cufftplanmany
	    hipfftPlanMany(&many_plan, 1, dimension_size, 0, 0, 0, 0, 0, 0, HIPFFT_Z2Z, 50);


	    printf("\n[Popwi::popprotadespread]: init deconvolve complete \n");
	}


	//Free all the memory that we allocated
	//TODO: check that this is comprehensive
	void cleanup() {
	  hipfftDestroy(plan1);
	  hipfftDestroy(plan2);
	  hipfftDestroy(many_plan);
	  checkCudaErrors(hipFree(d_dataa));
	  checkCudaErrors(hipFree(d_datab));
	  checkCudaErrors(hipFree(d_datac));
	  checkCudaErrors(hipFree(d_datad));
	}


	void gpu_rolling_dot_product(popComplex *in, popComplex *cfc, popComplex *out, int len, int fbins)
	{
		// TODO: better refactor thread and block sizes for any possible spreading code and fbin lengths
		rolling_scalar_multiply<<<fbins * 16, len / 16>>>(in, cfc, out, len);
		hipDeviceSynchronize();
	}


	// this is the functor which calculates magnitude's for samples in the keep zone
	// and calculates 0.0 for samples outside of the zone
	// note for some weird reason if this struct has a normal style constructor other basic CUDA functions are affected and refuse to run!??
	struct indexed_magnitude_squared_functor_fixed : public thrust::binary_function<int,popComplex,double>
		{
		public:
			int m_len;

			__host__ __device__
			double operator()(const int& index, const popComplex& a) const {

				int b = index % m_len; // fft bin

				// if we in the region we want to cutoff, return 0.0 for the magnitude
				if( (b > (m_len / 4)) && (b <= (3 * m_len /4)) )
				{
					return 0.0;
				}

				return a.x * a.x + a.y * a.y;
			}
		};

	void thrust_peak_detection(popComplex* in, thrust::device_vector<double>* d_mag_vec, double* peak, int* index, int len, int fbins)
	{
		int totalLen = len*fbins;

		// grab an iterator to the beginning of the data that was already cuda memcopied onto the gpu
		thrust::device_ptr<popComplex> d_vec_begin = thrust::device_pointer_cast(in);

//		// transform between two vectors like this:
//		// http://thrust.github.io/doc/group__transformations.html#ga68a3ba7d332887f1332ca3bc04453792

		indexed_magnitude_squared_functor_fixed functor = indexed_magnitude_squared_functor_fixed();
		functor.m_len = len;

		// this function is weird because it takes begin1, end1, begin2 but not end2.  so therefore end2 is calculated based on begin/end 1
		// the 4th argument is the beginning of the output, and the 5th is the functor

		// takes about 42000us
		thrust::transform(
				thrust::make_counting_iterator(0),
				thrust::make_counting_iterator(totalLen),
				d_vec_begin,
				d_mag_vec->begin(),
				functor);


		// find the maximum element using the gpu, and return a pointer to it (a device_vector::iterator)
		// this takes about 36000us
		thrust::device_vector<double>::iterator d_max_element_itr = thrust::max_element(d_mag_vec->begin(), d_mag_vec->end());

		unsigned int position = d_max_element_itr - d_mag_vec->begin();
		double max_val = *d_max_element_itr;
		*peak = max_val;
	}



}
