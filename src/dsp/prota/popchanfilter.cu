#include "hip/hip_runtime.h"
/******************************************************************************
* Copyright 2013 PopWi Technology Group, Inc. (PTG)
*
* This file is proprietary and exclusively owned by PTG or its associates.
* This document is protected by international and domestic patents where
* applicable. All rights reserved.
*
******************************************************************************/

//#include <hip/hip_complex.h>
#include <complex>
#include <iostream>
#include <stdexcept>
#include <cstdio>
#include <cmath>
#include "dsp/utils.hpp"

#include <hipfft/hipfft.h>

using namespace std;


__device__ float magnitude2( hipComplex& in )
{
	return in.x * in.x + in.y * in.y;
}

__device__ hipComplex operator*(const hipComplex& a, const hipComplex& b)
{
	hipComplex r;
	r.x = b.x*a.x - b.y*a.y;
	r.y = b.y*a.x + b.x*a.y;
	return r;
}

__device__ hipComplex operator+(const hipComplex& a, const hipComplex& b)
{
	hipComplex r;
	r.x = a.x + b.x;
	r.y = a.y + b.y;
	return r;
}

__global__ void rolling_scalar_multiply(hipComplex *in, hipComplex *cfc, hipComplex *out, int len)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int I = blockDim.x * gridDim.x;
	
	int fsearchbin = (I / len);
	int fidx = (i / len) - (fsearchbin / 2); // frequency modulation index
	int b = i % len; // fft bin
	int cidx = (b + fidx + len) % len;

	out[i] = in[b] * cfc[cidx];
}

__device__ unsigned IFloatFlip(unsigned f)
{
	unsigned mask = ((f >> 31) - 1) | 0x80000000;
	return f ^ mask;
}

__device__ unsigned FloatFlip(unsigned f)
{
	unsigned mask = -signed(f >> 31) | 0x80000000;
	return f ^ mask;
}

__global__ void peak_detection(hipComplex *in, float *peak, int len)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int I = blockDim.x * gridDim.x;

	int F = (I / len);
	int f = (i / len) - (F / 2); // frequency
	int b = i % len; // fft bin
	float mag; // magnitude of peak
	unsigned si; // sortable integer

	// don't look for peaks in padding
	if( b < (len / 4) ) return;
	if( b >= (3 * len / 4) ) return;

	// take the magnitude of the detection
	mag = magnitude2(in[i]);

	// transform into sortable integer
	// https://devtalk.nvidia.com/default/topic/406770/cuda-programming-and-performance/atomicmax-for-float/
	//si = *((unsigned*)&mag) ^ (-signed(*((unsigned*)&mag)>>31) | 0x80000000);
	si = FloatFlip((unsigned&)mag);

	// check to see if this is the highest recorded value
	atomicMax((unsigned*)peak, si);
}


extern "C"
{	
	hipComplex* d_dataa;
	hipComplex* d_datab;
	hipComplex* d_datac;
	hipComplex* d_datad;
	hipfftHandle plan1;
	hipfftHandle plan2;
	size_t g_len_chan; ///< length of time series in samples
	size_t g_len_fft; ///< length of fft in samples
	size_t g_start_chan = 16128;


	size_t gpu_channel_split(const complex<float> *h_data, complex<float> *out)
	{
		//double ch_start, ch_end, ch_ctr;

/*		ch_start = 903626953 + (3200000 / (double)g_len_fft * (double)g_start_chan) - 1600000;
		ch_end = 903626953 + (3200000 / (double)g_len_fft * ((double)g_start_chan + 1040)) - 1600000;
		ch_ctr = (ch_start + ch_end) / 2.0;*/
		//printf("channel start: %f (%llu), end: %f, ctr: %f\r\n", ch_start, g_start_chan, ch_end, ch_ctr);

		// shift zero-frequency component to center of spectrum
		unsigned small_bin_start = (g_start_chan + (g_len_fft/2)) % g_len_fft;

		// calculate small bin side-band size
		unsigned small_bin_sideband = g_len_chan / 2;

		// copy new host data into device memory
		hipMemcpy(d_dataa, h_data, g_len_fft * sizeof(hipComplex), hipMemcpyHostToDevice);
		hipDeviceSynchronize();

		// perform FFT on spectrum
		hipfftExecC2C(plan1, (hipfftComplex*)d_dataa, (hipfftComplex*)d_datab, HIPFFT_FORWARD);
		hipDeviceSynchronize();

		
		// chop spectrum up into 50 spreading channels low side-band
		hipMemcpy(d_datac,
			       d_datab + small_bin_start + small_bin_sideband,
			       small_bin_sideband * sizeof(hipComplex),
			       hipMemcpyDeviceToDevice);
		// chop spectrum up into 50 spreading channels high side-band
		hipMemcpy(d_datac + small_bin_sideband,
			       d_datab + small_bin_start,
			       small_bin_sideband * sizeof(hipComplex),
			       hipMemcpyDeviceToDevice);
		hipDeviceSynchronize();


		// put back into time domain
		hipfftExecC2C(plan2, (hipfftComplex*)d_datac, (hipfftComplex*)d_datad, HIPFFT_BACKWARD);
		hipDeviceSynchronize();
  		checkCudaErrors(hipGetLastError());

  		// Copy results to host
		hipMemcpy(out, d_datad, g_len_chan * sizeof(hipComplex), hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		
  		return 0;
	}


	void init_deconvolve(size_t len_fft, size_t len_chan)
	{
		g_len_chan = len_chan;
		g_len_fft = len_fft;

		// allocate CUDA memory
		checkCudaErrors(hipMalloc(&d_dataa, g_len_fft * sizeof(hipComplex)));
		checkCudaErrors(hipMalloc(&d_datab, g_len_fft * sizeof(hipComplex)));
		checkCudaErrors(hipMalloc(&d_datac, g_len_fft * sizeof(hipComplex)));
		checkCudaErrors(hipMalloc(&d_datad, g_len_fft * sizeof(hipComplex)));

		// initialize CUDA memory
		checkCudaErrors(hipMemset(d_dataa, 0, g_len_fft * sizeof(hipComplex)));
		checkCudaErrors(hipMemset(d_datab, 0, g_len_fft * sizeof(hipComplex)));
		checkCudaErrors(hipMemset(d_datac, 0, g_len_fft * sizeof(hipComplex)));
		checkCudaErrors(hipMemset(d_datad, 0, g_len_fft * sizeof(hipComplex)));

	    // setup FFT plans
	    hipfftPlan1d(&plan1, g_len_fft, HIPFFT_C2C, 1);
	    hipfftPlan1d(&plan2, g_len_chan, HIPFFT_C2C, 1);

	    printf("\n[Popwi::popprotadespread]: init deconvolve complete \n");
	}


	//Free all the memory that we allocated
	//TODO: check that this is comprehensive
	void cleanup() {
	  hipfftDestroy(plan1);
	  hipfftDestroy(plan2);
	  checkCudaErrors(hipFree(d_dataa));
	  checkCudaErrors(hipFree(d_datab));
	  checkCudaErrors(hipFree(d_datac));
	  checkCudaErrors(hipFree(d_datad));
	}


	void gpu_rolling_dot_product(hipComplex *in, hipComplex *cfc, hipComplex *out, int len, int fbins)
	{
		// TODO: better refactor thread and block sizes for any possible spreading code and fbin lengths
		rolling_scalar_multiply<<<fbins * 2, len / 2>>>(in, cfc, out, len);
		hipDeviceSynchronize();
	}

	void gpu_peak_detection(hipComplex* in, float* peak, int len, int fbins)
	{
		// TODO: better refactor thread and block sizes for any possible spreading code and fbin lengths
		peak_detection<<<fbins * 2, len / 2>>>(in, peak, len);
		hipDeviceSynchronize();
	}

}
