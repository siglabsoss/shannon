#include "hip/hip_runtime.h"
/******************************************************************************
* Copyright 2013 PopWi Technology Group, Inc. (PTG)
*
* This file is proprietary and exclusively owned by PTG or its associates.
* This document is protected by international and domestic patents where
* applicable. All rights reserved.
*
******************************************************************************/

// Bad practice, but I can't get multiple cuda files to link http://stackoverflow.com/questions/13683575/cuda-5-0-separate-compilation-of-library-with-cmake
#include <dsp/prota/popdeconvolve.cu>


#include <iostream>
#include <stdexcept>
#include <cstdio>
#include <cmath>
#include "dsp/utils.hpp"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/tuple.h>
#include <thrust/reduce.h>
#include <thrust/fill.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <cstdlib>
#include <time.h>

#include <dsp/prota/popchanfilter.cuh>



using namespace std;
using namespace thrust;

__device__ float magnitude2( hipComplex& in )
{
	return in.x * in.x + in.y * in.y;
}

__device__ hipComplex operator*(const hipComplex& a, const hipComplex& b)
{
	hipComplex r;
	r.x = b.x*a.x - b.y*a.y;
	r.y = b.y*a.x + b.x*a.y;
	return r;
}

__device__ hipComplex operator+(const hipComplex& a, const hipComplex& b)
{
	hipComplex r;
	r.x = a.x + b.x;
	r.y = a.y + b.y;
	return r;
}

__global__ void rolling_scalar_multiply(hipComplex *in, hipComplex *cfc, hipComplex *out, int len)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int I = blockDim.x * gridDim.x;
	
	int fsearchbin = (I / len);
	int fidx = (i / len) - (fsearchbin / 2); // frequency modulation index
	int b = i % len; // fft bin
	int cidx = (b + fidx + len) % len;

	out[i] = in[b] * cfc[cidx];
}

__device__ unsigned IFloatFlip(unsigned f)
{
	unsigned mask = ((f >> 31) - 1) | 0x80000000;
	return f ^ mask;
}

__device__ unsigned FloatFlip(unsigned f)
{
	unsigned mask = -signed(f >> 31) | 0x80000000;
	return f ^ mask;
}

__global__ void peak_detection(hipComplex *in, float *peak, int len)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int I = blockDim.x * gridDim.x;

	int F = (I / len);
	int f = (i / len) - (F / 2); // frequency
	int b = i % len; // fft bin
	float mag; // magnitude of peak
	unsigned si; // sortable integer

	// don't look for peaks in padding
	if( (b > (len / 4)) && (b <= (3 * len /4)) ) return;

	// take the magnitude of the detection
	mag = magnitude2(in[i]);

	// transform into sortable integer
	// https://devtalk.nvidia.com/default/topic/406770/cuda-programming-and-performance/atomicmax-for-float/
	//si = *((unsigned*)&mag) ^ (-signed(*((unsigned*)&mag)>>31) | 0x80000000);
	si = FloatFlip((unsigned&)mag);

	// check to see if this is the highest recorded value
	atomicMax((unsigned*)peak, si);
}


extern "C"
{	
	hipComplex* d_dataa;
	hipComplex* d_datab;
	hipComplex* d_datac;
	hipComplex* d_datad;
	hipfftHandle plan1;
	hipfftHandle plan2;
	size_t g_len_chan; ///< length of time series in samples
	size_t g_len_fft; ///< length of fft in samples
	size_t g_start_chan = 16128;


	size_t gpu_channel_split(const complex<float> *h_data, complex<float> *out)
	{
		//double ch_start, ch_end, ch_ctr;

/*		ch_start = 903626953 + (3200000 / (double)g_len_fft * (double)g_start_chan) - 1600000;
		ch_end = 903626953 + (3200000 / (double)g_len_fft * ((double)g_start_chan + 1040)) - 1600000;
		ch_ctr = (ch_start + ch_end) / 2.0;*/
		//printf("channel start: %f (%llu), end: %f, ctr: %f\r\n", ch_start, g_start_chan, ch_end, ch_ctr);

		// shift zero-frequency component to center of spectrum
		unsigned small_bin_start = (g_start_chan + (g_len_fft/2)) % g_len_fft;

		// calculate small bin side-band size
		unsigned small_bin_sideband = g_len_chan / 2;

		// copy new host data into device memory
		hipMemcpy(d_dataa, h_data, g_len_fft * sizeof(hipComplex), hipMemcpyHostToDevice);
		hipDeviceSynchronize();

		// perform FFT on spectrum
		hipfftExecC2C(plan1, (hipfftComplex*)d_dataa, (hipfftComplex*)d_datab, HIPFFT_FORWARD);
		hipDeviceSynchronize();

		
		// chop spectrum up into 50 spreading channels low side-band
		hipMemcpy(d_datac,
			       d_datab + small_bin_start + small_bin_sideband,
			       small_bin_sideband * sizeof(hipComplex),
			       hipMemcpyDeviceToDevice);
		// chop spectrum up into 50 spreading channels high side-band
		hipMemcpy(d_datac + small_bin_sideband,
			       d_datab + small_bin_start,
			       small_bin_sideband * sizeof(hipComplex),
			       hipMemcpyDeviceToDevice);
		hipDeviceSynchronize();


		// put back into time domain
		hipfftExecC2C(plan2, (hipfftComplex*)d_datac, (hipfftComplex*)d_datad, HIPFFT_BACKWARD);
		hipDeviceSynchronize();
  		checkCudaErrors(hipGetLastError());

  		// Copy results to host
		hipMemcpy(out, d_datad, g_len_chan * sizeof(hipComplex), hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		
  		return 0;
	}


	void init_deconvolve(size_t len_fft, size_t len_chan)
	{
		g_len_chan = len_chan;
		g_len_fft = len_fft;

		// allocate CUDA memory
		checkCudaErrors(hipMalloc(&d_dataa, g_len_fft * sizeof(hipComplex)));
		checkCudaErrors(hipMalloc(&d_datab, g_len_fft * sizeof(hipComplex)));
		checkCudaErrors(hipMalloc(&d_datac, g_len_fft * sizeof(hipComplex)));
		checkCudaErrors(hipMalloc(&d_datad, g_len_fft * sizeof(hipComplex)));

		// initialize CUDA memory
		checkCudaErrors(hipMemset(d_dataa, 0, g_len_fft * sizeof(hipComplex)));
		checkCudaErrors(hipMemset(d_datab, 0, g_len_fft * sizeof(hipComplex)));
		checkCudaErrors(hipMemset(d_datac, 0, g_len_fft * sizeof(hipComplex)));
		checkCudaErrors(hipMemset(d_datad, 0, g_len_fft * sizeof(hipComplex)));

	    // setup FFT plans
	    hipfftPlan1d(&plan1, g_len_fft, HIPFFT_C2C, 1);
	    hipfftPlan1d(&plan2, g_len_chan, HIPFFT_C2C, 1);

	    printf("\n[Popwi::popprotadespread]: init deconvolve complete \n");
	}


	//Free all the memory that we allocated
	//TODO: check that this is comprehensive
	void cleanup() {
	  hipfftDestroy(plan1);
	  hipfftDestroy(plan2);
	  checkCudaErrors(hipFree(d_dataa));
	  checkCudaErrors(hipFree(d_datab));
	  checkCudaErrors(hipFree(d_datac));
	  checkCudaErrors(hipFree(d_datad));
	}


	void gpu_rolling_dot_product(hipComplex *in, hipComplex *cfc, hipComplex *out, int len, int fbins)
	{
		// TODO: better refactor thread and block sizes for any possible spreading code and fbin lengths
		rolling_scalar_multiply<<<fbins * 16, len / 16>>>(in, cfc, out, len);
		hipDeviceSynchronize();
	}

	void gpu_peak_detection(hipComplex* in, float* peak, int len, int fbins)
	{
		// TODO: better refactor thread and block sizes for any possible spreading code and fbin lengths
		peak_detection<<<fbins * 16, len / 16>>>(in, peak, len);
		hipDeviceSynchronize();
	}


	// this is the functor which calculates magnitude's for samples in the keep zone
	// and calculates 0.0 for samples outside of the zone
	// note for some weird reason if this struct has a normal style constructor other basic CUDA functions are affected and refuse to run!??
	struct indexed_magnitude_squared_functor_fixed : public thrust::binary_function<int,hipComplex,float>
		{
		public:
			int m_len;

			__host__ __device__
			float operator()(const int& index, const hipComplex& a) const {

				int b = index % m_len; // fft bin

				// if we in the region we want to cutoff, return 0.0 for the magnitude
				if( (b > (m_len / 4)) && (b <= (3 * m_len /4)) )
				{
					return 0.0;
				}

				return a.x * a.x + a.y * a.y;
			}
		};

	void thrust_peak_detection(hipComplex* in, thrust::device_vector<float>* d_mag_vec, float* peak, int* index, int len, int fbins)
	{
		int totalLen = len*fbins;

		// grab an iterator to the beginning of the data that was already cuda memcopied onto the gpu
		thrust::device_ptr<hipComplex> d_vec_begin = thrust::device_pointer_cast(in);

//		// transform between two vectors like this:
//		// http://thrust.github.io/doc/group__transformations.html#ga68a3ba7d332887f1332ca3bc04453792

		indexed_magnitude_squared_functor_fixed functor = indexed_magnitude_squared_functor_fixed();
		functor.m_len = len;

		// this function is weird because it takes begin1, end1, begin2 but not end2.  so therefore end2 is calculated based on begin/end 1
		// the 4th argument is the beginning of the output, and the 5th is the functor

		// takes about 42000us
		thrust::transform(
				thrust::make_counting_iterator(0),
				thrust::make_counting_iterator(totalLen),
				d_vec_begin,
				d_mag_vec->begin(),
				functor);


		// find the maximum element using the gpu, and return a pointer to it (a device_vector::iterator)
		// this takes about 36000us
		thrust::device_vector<float>::iterator d_max_element_itr = thrust::max_element(d_mag_vec->begin(), d_mag_vec->end());

		unsigned int position = d_max_element_itr - d_mag_vec->begin();
		float max_val = *d_max_element_itr;
		*peak = max_val;
	}



}
