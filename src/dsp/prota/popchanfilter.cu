#include "hip/hip_runtime.h"
/******************************************************************************
* Copyright 2013 PopWi Technology Group, Inc. (PTG)
*
* This file is proprietary and exclusively owned by PTG or its associates.
* This document is protected by international and domestic patents where
* applicable. All rights reserved.
*
******************************************************************************/

//#include <hip/hip_complex.h>
#include <complex>
#include <iostream>
#include <stdexcept>
#include <cstdio>
#include <cmath>
#include "dsp/utils.hpp"

#include <hipfft/hipfft.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/tuple.h>
#include <thrust/reduce.h>
#include <thrust/fill.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <cstdlib>
#include <time.h>

using namespace std;
using namespace thrust;

__device__ float magnitude2( hipComplex& in )
{
	return in.x * in.x + in.y * in.y;
}

__device__ hipComplex operator*(const hipComplex& a, const hipComplex& b)
{
	hipComplex r;
	r.x = b.x*a.x - b.y*a.y;
	r.y = b.y*a.x + b.x*a.y;
	return r;
}

__device__ hipComplex operator+(const hipComplex& a, const hipComplex& b)
{
	hipComplex r;
	r.x = a.x + b.x;
	r.y = a.y + b.y;
	return r;
}

__global__ void rolling_scalar_multiply(hipComplex *in, hipComplex *cfc, hipComplex *out, int len)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int I = blockDim.x * gridDim.x;
	
	int fsearchbin = (I / len);
	int fidx = (i / len) - (fsearchbin / 2); // frequency modulation index
	int b = i % len; // fft bin
	int cidx = (b + fidx + len) % len;

	out[i] = in[b] * cfc[cidx];
}

__device__ unsigned IFloatFlip(unsigned f)
{
	unsigned mask = ((f >> 31) - 1) | 0x80000000;
	return f ^ mask;
}

__device__ unsigned FloatFlip(unsigned f)
{
	unsigned mask = -signed(f >> 31) | 0x80000000;
	return f ^ mask;
}

__global__ void peak_detection(hipComplex *in, float *peak, int len)
{

	int block = blockIdx.x;
	int blockDimx = blockDim.x;
	int threadIdxx = threadIdx.x;

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int I = blockDim.x * gridDim.x;

	int F = (I / len);
	int f = (i / len) - (F / 2); // frequency
	int b = i % len; // fft bin





	float mag; // magnitude of peak
	unsigned si; // sortable integer

	// don't look for peaks in padding
	if( (b > (len / 4)) && (b <= (3 * len /4)) ) return;

	// take the magnitude of the detection
	mag = magnitude2(in[i]);

	// transform into sortable integer
	// https://devtalk.nvidia.com/default/topic/406770/cuda-programming-and-performance/atomicmax-for-float/
	//si = *((unsigned*)&mag) ^ (-signed(*((unsigned*)&mag)>>31) | 0x80000000);
	si = FloatFlip((unsigned&)mag);

	// check to see if this is the highest recorded value
	atomicMax((unsigned*)peak, si);
}

template <class T>
struct bigger_magnitude_tuple {
    __device__ __host__
    tuple<T,int> operator()(const tuple<T,int> &a, const tuple<T,int> &b)
    {

    	float maga = ( get<0>(a).x * get<0>(a).x ) + ( get<0>(a).y * get<0>(a).y );
    	float magb = ( get<0>(b).x * get<0>(b).x ) + ( get<0>(b).y * get<0>(b).y );


        if (maga > magb) return a;
        else return b;
//
//
//        if (get<0>(a) > get<0>(b)) return a;
//        else return b;


//        return a;

    }

};


// return the biggest of two tuples
template <class T>
struct bigger_tuple {
    __device__ __host__
    tuple<T,int> operator()(const tuple<T,int> &a, const tuple<T,int> &b)
    {
        if (get<0>(a) > get<0>(b)) return a;
        else return b;
    }

};



template <class T>
int max_index(device_vector<T>& vec) {
//
    // create implicit index sequence [0, 1, 2, ... )
    counting_iterator<int> begin(0); counting_iterator<int> end(vec.size());
    tuple<T,int> init(vec[0],0);
    tuple<T,int> smallest;

    smallest = reduce(make_zip_iterator(make_tuple(vec.begin(), begin)), make_zip_iterator(make_tuple(vec.end(), end)),
                      init, bigger_tuple<T>());
    return get<1>(smallest);
}





template <class T>
int max_magnitude_index(device_vector<T>& vec) {
	    // create implicit index sequence [0, 1, 2, ... )
	    counting_iterator<int> begin(0); counting_iterator<int> end(vec.size());
	    tuple<T,int> init(vec[0],0);
	    tuple<T,int> smallest;

	    smallest = reduce(make_zip_iterator(make_tuple(vec.begin(), begin)), make_zip_iterator(make_tuple(vec.end(), end)),
	                      init, bigger_magnitude_tuple<T>());
	    return get<1>(smallest);
}




extern "C"
{	
	hipComplex* d_dataa;
	hipComplex* d_datab;
	hipComplex* d_datac;
	hipComplex* d_datad;
	hipfftHandle plan1;
	hipfftHandle plan2;
	size_t g_len_chan; ///< length of time series in samples
	size_t g_len_fft; ///< length of fft in samples
	size_t g_start_chan = 16128;


	size_t gpu_channel_split(const complex<float> *h_data, complex<float> *out)
	{
		//double ch_start, ch_end, ch_ctr;

/*		ch_start = 903626953 + (3200000 / (double)g_len_fft * (double)g_start_chan) - 1600000;
		ch_end = 903626953 + (3200000 / (double)g_len_fft * ((double)g_start_chan + 1040)) - 1600000;
		ch_ctr = (ch_start + ch_end) / 2.0;*/
		//printf("channel start: %f (%llu), end: %f, ctr: %f\r\n", ch_start, g_start_chan, ch_end, ch_ctr);

		// shift zero-frequency component to center of spectrum
		unsigned small_bin_start = (g_start_chan + (g_len_fft/2)) % g_len_fft;

		// calculate small bin side-band size
		unsigned small_bin_sideband = g_len_chan / 2;

		// copy new host data into device memory
		hipMemcpy(d_dataa, h_data, g_len_fft * sizeof(hipComplex), hipMemcpyHostToDevice);
		hipDeviceSynchronize();

		// perform FFT on spectrum
		hipfftExecC2C(plan1, (hipfftComplex*)d_dataa, (hipfftComplex*)d_datab, HIPFFT_FORWARD);
		hipDeviceSynchronize();

		
		// chop spectrum up into 50 spreading channels low side-band
		hipMemcpy(d_datac,
			       d_datab + small_bin_start + small_bin_sideband,
			       small_bin_sideband * sizeof(hipComplex),
			       hipMemcpyDeviceToDevice);
		// chop spectrum up into 50 spreading channels high side-band
		hipMemcpy(d_datac + small_bin_sideband,
			       d_datab + small_bin_start,
			       small_bin_sideband * sizeof(hipComplex),
			       hipMemcpyDeviceToDevice);
		hipDeviceSynchronize();


		// put back into time domain
		hipfftExecC2C(plan2, (hipfftComplex*)d_datac, (hipfftComplex*)d_datad, HIPFFT_BACKWARD);
		hipDeviceSynchronize();
  		checkCudaErrors(hipGetLastError());

  		// Copy results to host
		hipMemcpy(out, d_datad, g_len_chan * sizeof(hipComplex), hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		
  		return 0;
	}


	void init_deconvolve(size_t len_fft, size_t len_chan)
	{
		g_len_chan = len_chan;
		g_len_fft = len_fft;

		// allocate CUDA memory
		checkCudaErrors(hipMalloc(&d_dataa, g_len_fft * sizeof(hipComplex)));
		checkCudaErrors(hipMalloc(&d_datab, g_len_fft * sizeof(hipComplex)));
		checkCudaErrors(hipMalloc(&d_datac, g_len_fft * sizeof(hipComplex)));
		checkCudaErrors(hipMalloc(&d_datad, g_len_fft * sizeof(hipComplex)));

		// initialize CUDA memory
		checkCudaErrors(hipMemset(d_dataa, 0, g_len_fft * sizeof(hipComplex)));
		checkCudaErrors(hipMemset(d_datab, 0, g_len_fft * sizeof(hipComplex)));
		checkCudaErrors(hipMemset(d_datac, 0, g_len_fft * sizeof(hipComplex)));
		checkCudaErrors(hipMemset(d_datad, 0, g_len_fft * sizeof(hipComplex)));

	    // setup FFT plans
	    hipfftPlan1d(&plan1, g_len_fft, HIPFFT_C2C, 1);
	    hipfftPlan1d(&plan2, g_len_chan, HIPFFT_C2C, 1);

	    printf("\n[Popwi::popprotadespread]: init deconvolve complete \n");
	}


	//Free all the memory that we allocated
	//TODO: check that this is comprehensive
	void cleanup() {
	  hipfftDestroy(plan1);
	  hipfftDestroy(plan2);
	  checkCudaErrors(hipFree(d_dataa));
	  checkCudaErrors(hipFree(d_datab));
	  checkCudaErrors(hipFree(d_datac));
	  checkCudaErrors(hipFree(d_datad));
	}


	void gpu_rolling_dot_product(hipComplex *in, hipComplex *cfc, hipComplex *out, int len, int fbins)
	{
		// TODO: better refactor thread and block sizes for any possible spreading code and fbin lengths
		rolling_scalar_multiply<<<fbins * 16, len / 16>>>(in, cfc, out, len);
		hipDeviceSynchronize();
	}

	void gpu_peak_detection(hipComplex* in, float* peak, int len, int fbins)
	{
		// TODO: better refactor thread and block sizes for any possible spreading code and fbin lengths
		peak_detection<<<fbins * 16, len / 16>>>(in, peak, len);
		hipDeviceSynchronize();
	}

	struct magnitude_squared_functor
	{
		__host__ __device__
		float operator()(const hipComplex& a) const {
			return a.x * a.x + a.y * a.y;
		}
	};


	void thrust_peak_detection(hipComplex* in, float* peak, int* index, int len, int fbins)
	{

		// allocate device_vector with len elements to hold the magnitude
		thrust::device_vector<float> d_mag_vec(len);


		// static cast pointer so we can use hipComplex types
//		const hipComplex* cuIn = (hipComplex*)in;


		//		thrust::device_vector<hipComplex> d_vecc(inCuComplex, inCuComplex+len);

		// transfer data to the device
//		thrust::device_vector< hipComplex > d_vec(cuIn, cuIn+len);


		thrust::device_ptr<hipComplex> d_vec_begin = thrust::device_pointer_cast(in);



//
//		//
//		cout << endl << endl;
//
//		for(int i = 0; i < d_vec.size(); i++)
//		{
//			hipComplex copy = d_vec[i];
//
//			std::cout << "d_vec[" << i << "] = " << copy.x << ", " << copy.y << std::endl;
//		}
//
//		cout << endl << endl;

		//
		//
		//		// compute magnitude from d_vec and store in d_mag_vec
		thrust::transform(d_vec_begin, (d_vec_begin + len), d_mag_vec.begin(), magnitude_squared_functor());
		//
		//
		//		 for(int i = 0; i < d_mag_vec.size(); i++)
		//		        std::cout << "d_mag_vec[" << i << "] = " << d_mag_vec[i] << std::endl;
		//
		//		 cout << endl << endl << endl << endl << endl << endl;

		thrust::device_vector<float>::iterator d_max_element_itr = thrust::max_element(d_mag_vec.begin(), d_mag_vec.end());

		unsigned int position = d_max_element_itr - d_mag_vec.begin();
		float max_val = *d_max_element_itr;

		std::cout << "The maximum value is " << max_val << " at position " << position << std::endl;

		cout << endl << endl << endl << endl << endl << endl;


		//		int index = max_magnitude_index(d_vec);
		//
		////		int index = max_index(test);
		//
		//
		//
		//		std::cout << std::endl << std::endl << std::endl << "Max index is:" << index << std::endl;
		//		std::cout << "Value is: " << in[index] <<std::endl << endl << endl;

		//		int crash = 0/0;

	}










//	void thrust_peak_detection(const complex<float>* in, float* peak, int len, int fbins)
//	{
//
//		// allocate device_vector with len elements
//		thrust::device_vector<float> d_mag_vec(len);
//
//
//		// static cast pointer so we can use hipComplex types
//		const hipComplex* cuIn = (hipComplex*)in;
//
//
//		//		thrust::device_vector<hipComplex> d_vecc(inCuComplex, inCuComplex+len);
//
//		// transfer data to the device
//		thrust::device_vector< hipComplex > d_vec(cuIn, cuIn+len);
//		//
//		cout << endl << endl;
//
//		for(int i = 0; i < d_vec.size(); i++)
//		{
//			hipComplex copy = d_vec[i];
//
//			std::cout << "d_vec[" << i << "] = " << copy.x << ", " << copy.y << std::endl;
//		}
//
//		cout << endl << endl;
//
//		//
//		//
//		//		// compute magnitude from d_vec and store in d_mag_vec
//		thrust::transform(d_vec.begin(), d_vec.end(), d_mag_vec.begin(), magnitude_squared_functor());
//		//
//		//
//		//		 for(int i = 0; i < d_mag_vec.size(); i++)
//		//		        std::cout << "d_mag_vec[" << i << "] = " << d_mag_vec[i] << std::endl;
//		//
//		//		 cout << endl << endl << endl << endl << endl << endl;
//
//		thrust::device_vector<float>::iterator d_max_element_itr = thrust::max_element(d_mag_vec.begin(), d_mag_vec.end());
//
//		unsigned int position = d_max_element_itr - d_mag_vec.begin();
//		float max_val = *d_max_element_itr;
//
//		std::cout << "The maximum value is " << max_val << " at position " << position << std::endl;
//
//		cout << endl << endl << endl << endl << endl << endl;
//
//
//		//		int index = max_magnitude_index(d_vec);
//		//
//		////		int index = max_index(test);
//		//
//		//
//		//
//		//		std::cout << std::endl << std::endl << std::endl << "Max index is:" << index << std::endl;
//		//		std::cout << "Value is: " << in[index] <<std::endl << endl << endl;
//
//		//		int crash = 0/0;
//
//	}








}
