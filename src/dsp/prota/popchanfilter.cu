#include "hip/hip_runtime.h"
/******************************************************************************
* Copyright 2013 PopWi Technology Group, Inc. (PTG)
*
* This file is proprietary and exclusively owned by PTG or its associates.
* This document is protected by international and domestic patents where
* applicable. All rights reserved.
*
******************************************************************************/

// Bad practice, but I can't get multiple cuda files to link http://stackoverflow.com/questions/13683575/cuda-5-0-separate-compilation-of-library-with-cmake
#include <dsp/prota/popdeconvolve.cu>
#include <dsp/common/poptypes.cu>


#include <iostream>
#include <stdexcept>
#include <cstdio>
#include <cmath>
#include "dsp/utils.hpp"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/tuple.h>
#include <thrust/reduce.h>
#include <thrust/fill.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <cstdlib>
#include <time.h>

#include <dsp/common/poptypes.cuh>

#include <dsp/prota/popchanfilter.cuh>

#include "core/basestationfreq.h"



using namespace std;
using namespace thrust;


__global__ void rolling_scalar_multiply(popComplex *in, popComplex *cfc, popComplex *out, int len)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int I = blockDim.x * gridDim.x;
	
	int fsearchbin = (I / len);
	int fidx = (i / len) - (fsearchbin / 2); // frequency modulation index
	int b = i % len; // fft bin
	int cidx = (b + fidx + len) % len;

	out[i] = in[b] * cfc[cidx];
}

__device__ unsigned IFloatFlip(unsigned f)
{
	unsigned mask = ((f >> 31) - 1) | 0x80000000;
	return f ^ mask;
}

__device__ unsigned FloatFlip(unsigned f)
{
	unsigned mask = -signed(f >> 31) | 0x80000000;
	return f ^ mask;
}



extern "C"
{	
	popComplex* d_dataa;
	popComplex* d_datab;
	popComplex* d_datac;
	popComplex* d_datad;
	hipfftHandle plan1;
	hipfftHandle plan2;
	size_t g_len_chan; ///< length of time series in samples
	size_t g_len_fft; ///< length of fft in samples
	size_t g_start_chan;


	size_t gpu_channel_split(const complex<double> *h_data, complex<double> *out)
	{
		//double ch_start, ch_end, ch_ctr;

		g_start_chan = bsf_channel_fbin_low(9);

/*		ch_start = 903626953 + (3200000 / (double)g_len_fft * (double)g_start_chan) - 1600000;
		ch_end = 903626953 + (3200000 / (double)g_len_fft * ((double)g_start_chan + 1040)) - 1600000;
		ch_ctr = (ch_start + ch_end) / 2.0;*/
		//printf("channel start: %f (%llu), end: %f, ctr: %f\r\n", ch_start, g_start_chan, ch_end, ch_ctr);

		// shift zero-frequency component to center of spectrum
		unsigned small_bin_start = (g_start_chan + (g_len_fft/2)) % g_len_fft;

		// calculate small bin side-band size
		unsigned small_bin_sideband = g_len_chan / 2;

		// copy new host data into device memory
		hipMemcpy(d_dataa, h_data, g_len_fft * sizeof(popComplex), hipMemcpyHostToDevice);
		hipDeviceSynchronize();

		// perform FFT on spectrum
		hipfftExecZ2Z(plan1, (hipfftDoubleComplex*)d_dataa, (hipfftDoubleComplex*)d_datab, HIPFFT_FORWARD);
		hipDeviceSynchronize();

		
		// chop spectrum up into 50 spreading channels low side-band
		hipMemcpy(d_datac,
			       d_datab + small_bin_start + small_bin_sideband,
			       small_bin_sideband * sizeof(popComplex),
			       hipMemcpyDeviceToDevice);
		// chop spectrum up into 50 spreading channels high side-band
		hipMemcpy(d_datac + small_bin_sideband,
			       d_datab + small_bin_start,
			       small_bin_sideband * sizeof(popComplex),
			       hipMemcpyDeviceToDevice);
		hipDeviceSynchronize();


		// put back into time domain
		hipfftExecZ2Z(plan2, (hipfftDoubleComplex*)d_datac, (hipfftDoubleComplex*)d_datad, HIPFFT_BACKWARD);
		hipDeviceSynchronize();
  		checkCudaErrors(hipGetLastError());

  		// Copy results to host
		hipMemcpy(out, d_datad, g_len_chan * sizeof(popComplex), hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		
  		return 0;
	}


	void init_deconvolve(size_t len_fft, size_t len_chan)
	{
		g_len_chan = len_chan;
		g_len_fft = len_fft;

		// allocate CUDA memory
		checkCudaErrors(hipMalloc(&d_dataa, g_len_fft * sizeof(popComplex)));
		checkCudaErrors(hipMalloc(&d_datab, g_len_fft * sizeof(popComplex)));
		checkCudaErrors(hipMalloc(&d_datac, g_len_fft * sizeof(popComplex)));
		checkCudaErrors(hipMalloc(&d_datad, g_len_fft * sizeof(popComplex)));

		// initialize CUDA memory
		checkCudaErrors(hipMemset(d_dataa, 0, g_len_fft * sizeof(popComplex)));
		checkCudaErrors(hipMemset(d_datab, 0, g_len_fft * sizeof(popComplex)));
		checkCudaErrors(hipMemset(d_datac, 0, g_len_fft * sizeof(popComplex)));
		checkCudaErrors(hipMemset(d_datad, 0, g_len_fft * sizeof(popComplex)));

	    // setup FFT plans
	    hipfftPlan1d(&plan1, g_len_fft, HIPFFT_Z2Z, 1);
	    hipfftPlan1d(&plan2, g_len_chan, HIPFFT_Z2Z, 1);

	    printf("\n[Popwi::popprotadespread]: init deconvolve complete \n");
	}


	//Free all the memory that we allocated
	//TODO: check that this is comprehensive
	void cleanup() {
	  hipfftDestroy(plan1);
	  hipfftDestroy(plan2);
	  checkCudaErrors(hipFree(d_dataa));
	  checkCudaErrors(hipFree(d_datab));
	  checkCudaErrors(hipFree(d_datac));
	  checkCudaErrors(hipFree(d_datad));
	}


	void gpu_rolling_dot_product(popComplex *in, popComplex *cfc, popComplex *out, int len, int fbins)
	{
		// TODO: better refactor thread and block sizes for any possible spreading code and fbin lengths
		rolling_scalar_multiply<<<fbins * 16, len / 16>>>(in, cfc, out, len);
		hipDeviceSynchronize();
	}


	// this is the functor which calculates magnitude's for samples in the keep zone
	// and calculates 0.0 for samples outside of the zone
	// note for some weird reason if this struct has a normal style constructor other basic CUDA functions are affected and refuse to run!??
	struct indexed_magnitude_squared_functor_fixed : public thrust::binary_function<int,popComplex,double>
		{
		public:
			int m_len;

			__host__ __device__
			double operator()(const int& index, const popComplex& a) const {

				int b = index % m_len; // fft bin

				// if we in the region we want to cutoff, return 0.0 for the magnitude
				if( (b > (m_len / 4)) && (b <= (3 * m_len /4)) )
				{
					return 0.0;
				}

				return a.x * a.x + a.y * a.y;
			}
		};

	void thrust_peak_detection(popComplex* in, thrust::device_vector<double>* d_mag_vec, double* peak, int* index, int len, int fbins)
	{
		int totalLen = len*fbins;

		// grab an iterator to the beginning of the data that was already cuda memcopied onto the gpu
		thrust::device_ptr<popComplex> d_vec_begin = thrust::device_pointer_cast(in);

//		// transform between two vectors like this:
//		// http://thrust.github.io/doc/group__transformations.html#ga68a3ba7d332887f1332ca3bc04453792

		indexed_magnitude_squared_functor_fixed functor = indexed_magnitude_squared_functor_fixed();
		functor.m_len = len;

		// this function is weird because it takes begin1, end1, begin2 but not end2.  so therefore end2 is calculated based on begin/end 1
		// the 4th argument is the beginning of the output, and the 5th is the functor

		// takes about 42000us
		thrust::transform(
				thrust::make_counting_iterator(0),
				thrust::make_counting_iterator(totalLen),
				d_vec_begin,
				d_mag_vec->begin(),
				functor);


		// find the maximum element using the gpu, and return a pointer to it (a device_vector::iterator)
		// this takes about 36000us
		thrust::device_vector<double>::iterator d_max_element_itr = thrust::max_element(d_mag_vec->begin(), d_mag_vec->end());

		unsigned int position = d_max_element_itr - d_mag_vec->begin();
		double max_val = *d_max_element_itr;
		*peak = max_val;
	}



}
