#include "hip/hip_runtime.h"
/******************************************************************************
* Copyright 2013 PopWi Technology Group, Inc. (PTG)
*
* This file is proprietary and exclusively owned by PTG or its associates.
* This document is protected by international and domestic patents where
* applicable. All rights reserved.
*
******************************************************************************/

//#include <hip/hip_complex.h>
#include <complex>
#include <iostream>
#include <stdexcept>
#include "dsp/utils.hpp"

#include <boost/math/common_factor.hpp>

#include <hipfft/hipfft.h>

using namespace std;

#define MAX_THREADS_PER_BLOCK 1024
#define IFFT_PADDING_FACTOR 2


__device__ float magnitude2( hipComplex& in )
{
	return in.x * in.x + in.y * in.y;
}

__device__ hipComplex operator*(const hipComplex& a, const hipComplex& b)
{
	hipComplex r;
	r.x = b.x*a.x - b.y*a.y;
	r.y = b.y*a.x + b.x*a.y;
	return r;
}

__device__ hipComplex operator+(const hipComplex& a, const hipComplex& b)
{
	hipComplex r;
	r.x = a.x + b.x;
	r.y = a.y + b.y;
	return r;
}

#define PN_LEN 800
#define SHARED_MEMORY_STEPS 2


__global__ void deconvolve(hipComplex *pn, hipComplex *old, hipComplex *in, hipComplex *out, int pn_len)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int memIdx1;
	int n;

	hipComplex s;
	
	// shared memory size 48kB
	__shared__ hipComplex smem_data[PN_LEN * 2]; // 12,800 bytes
	hipComplex* smem_new_data_ptr = smem_data + PN_LEN;
	__shared__ hipComplex smem_pn[PN_LEN]; // 6,400 bytes = 19,200 total

	s.x = 0.0;
	s.y = 0.0;

	if(i >= pn_len) return;

	// copy old buffer into shared memory
	for( n = 0; n < gridDim.x; n++ )
	{
		memIdx1 = n * blockDim.x + threadIdx.x;
		smem_data[memIdx1] = old[memIdx1];
	}

	// copy new buffer into shared memory
	for( n = 0; n < gridDim.x; n++ )
	{
		memIdx1 = n * blockDim.x + threadIdx.x;
		smem_new_data_ptr[memIdx1] = in[memIdx1];
	}

	// copy PN code into shared memory
	for( n = 0; n < gridDim.x; n++ )
	{
		memIdx1 = n * blockDim.x + threadIdx.x;
		smem_pn[memIdx1] = pn[memIdx1];
	}

	// Must sync to ensure all data copied in
	__syncthreads();

	// // Perform deconvolutoin
	for(n = 0; n < pn_len; n++)
	{
	 	s = smem_data[n + i] * smem_pn[n] + s;
	}

	out[i] = s;

}

extern "C"
{	
	hipComplex *d_prncode;
	hipComplex *d_dataa;
	hipComplex *d_datab;
	hipComplex *d_datac;
	hipComplex *d_datad;
	hipComplex *d_datad_upper;
	hipComplex *d_datae;
	hipfftHandle plan1;
	hipfftHandle plan2;
	size_t h_len_chan; ///< length of time series in samples
	size_t h_len_chan_padded; ///< length of interpolated time series
	size_t h_len_fft; ///< length of fft in samples
	size_t h_len_pn;
	size_t h_decon_idx; ///< index of deconvolution operation


	size_t gpu_channel_split(const complex<float> *h_data)
	{
		// shift zero-frequency component to center of spectrum
		unsigned small_bin_start = (16059 + (h_len_fft/2)) % h_len_fft;;

		// calculate zero array size
		unsigned small_bin_padding = h_len_chan * (IFFT_PADDING_FACTOR-1);

		// calculate small bin side-band size
		unsigned small_bin_sideband = h_len_chan / 2;

		// copy new host data into device memory
		hipMemcpy(d_dataa, h_data, h_len_fft * sizeof(hipComplex), hipMemcpyHostToDevice);

		// perform FFT on spectrum
		hipfftExecC2C(plan1, (hipfftComplex*)d_dataa, (hipfftComplex*)d_datab, HIPFFT_FORWARD);
		hipDeviceSynchronize();

		
		// chop spectrum up into 50 spreading channels low side-band
		hipMemcpy(d_datac,
			       d_datab + small_bin_start + small_bin_sideband,
			       small_bin_sideband * sizeof(hipComplex),
			       hipMemcpyDeviceToDevice);
		// chop spectrum up into 50 spreading channels high side-band
		hipMemcpy(d_datac + small_bin_sideband + small_bin_padding,
			       d_datab + small_bin_start,
			       small_bin_sideband * sizeof(hipComplex),
			       hipMemcpyDeviceToDevice);

		// swap double buffer
		hipMemcpy(d_datad,
			       d_datad_upper,
			       h_len_chan_padded * sizeof(hipComplex),
			       hipMemcpyDeviceToDevice);
		hipDeviceSynchronize();
  		checkCudaErrors(hipGetLastError());

		// put back into time domain
		hipfftExecC2C(plan2, (hipfftComplex*)d_datac, (hipfftComplex*)d_datad_upper, HIPFFT_BACKWARD);
		hipDeviceSynchronize();
  		checkCudaErrors(hipGetLastError());
		
  		h_decon_idx += h_len_chan_padded;
  		return h_decon_idx;
	}

	size_t gpu_demod(complex<float> *out)
	{
		hipComplex* old_data = d_datad + h_len_chan_padded - h_decon_idx;
		hipComplex* new_data = old_data + h_len_pn;

  		// deconvolve PN codes
		deconvolve<<<1, 800>>>(d_prncode, old_data, new_data, d_datae, h_len_pn);
		hipDeviceSynchronize();
		
		// Copy [deconvolved] results to host
		hipMemcpy(out, d_datae, h_len_pn * sizeof(hipComplex), hipMemcpyDeviceToHost);

		h_decon_idx -= h_len_pn;
		return h_decon_idx;
	}


	void init_deconvolve(complex<float> *h_pn, size_t len_pn, size_t len_fft, size_t len_chan)
	{
		h_len_chan = len_chan;
		h_len_chan_padded = len_chan * IFFT_PADDING_FACTOR;
		h_len_fft = len_fft;
		h_len_pn = len_pn;
		h_decon_idx = 0;

		// allocate CUDA memory
		checkCudaErrors(hipMalloc(&d_prncode, h_len_pn * sizeof(hipComplex)));

		checkCudaErrors(hipMalloc(&d_dataa, h_len_fft * sizeof(hipComplex)));
		checkCudaErrors(hipMalloc(&d_datab, h_len_fft * sizeof(hipComplex)));
		checkCudaErrors(hipMalloc(&d_datac, h_len_chan_padded * sizeof(hipComplex)));
		checkCudaErrors(hipMalloc(&d_datad, 2 * h_len_chan_padded * sizeof(hipComplex))); // double buffered
		d_datad_upper = d_datad + h_len_chan_padded;
		checkCudaErrors(hipMalloc(&d_datae, h_len_pn * sizeof(hipComplex)));

		// initialize CUDA memory
		checkCudaErrors(hipMemcpy(d_prncode, h_pn, h_len_pn * sizeof(hipComplex), hipMemcpyHostToDevice));

		checkCudaErrors(hipMemset(d_dataa, 0, h_len_fft * sizeof(hipComplex)));
		checkCudaErrors(hipMemset(d_datab, 0, h_len_fft * sizeof(hipComplex)));
		checkCudaErrors(hipMemset(d_datac, 0, h_len_chan_padded * sizeof(hipComplex)));
		checkCudaErrors(hipMemset(d_datad, 0, 2 * h_len_chan_padded * sizeof(hipComplex))); // dobule buffered
		checkCudaErrors(hipMemset(d_datae, 0, h_len_pn * sizeof(hipComplex)));
		

	    // setup FFT plans
	    hipfftPlan1d(&plan1, h_len_fft, HIPFFT_C2C, 1);
	    hipfftPlan1d(&plan2, h_len_chan_padded, HIPFFT_C2C, 1);

	    printf("\n[Popwi::popprotadespread]: init deconvolve complete \n");
	}


	//Free all the memory that we allocated
	//TODO: check that this is comprehensive
	void cleanup() {
	  hipfftDestroy(plan1);
	  hipfftDestroy(plan2);
	  checkCudaErrors(hipFree(d_prncode));
	  checkCudaErrors(hipFree(d_dataa));
	  checkCudaErrors(hipFree(d_datab));
	  checkCudaErrors(hipFree(d_datac));
	  checkCudaErrors(hipFree(d_datad));
	  checkCudaErrors(hipFree(d_datae));
	}

}