#include "hip/hip_runtime.h"
/******************************************************************************
* Copyright 2013 PopWi Technology Group, Inc. (PTG)
*
* This file is proprietary and exclusively owned by PTG or its associates.
* This document is protected by international and domestic patents where
* applicable. All rights reserved.
*
******************************************************************************/

//#include <hip/hip_complex.h>
#include <complex>
#include <iostream>
#include <stdexcept>
#include "dsp/utils.hpp"

#include <hipfft/hipfft.h>

using namespace std;

#define MAX_THREADS_PER_BLOCK 1024
#define IFFT_PADDING_FACTOR 2


__device__ float magnitude2( hipComplex& in )
{
	return in.x * in.x + in.y * in.y;
}

__device__ hipComplex operator*(const hipComplex& a, const hipComplex& b)
{
	hipComplex r;
	r.x = b.x*a.x - b.y*a.y;
	r.y = b.y*a.x + b.x*a.y;
	return r;
}

__device__ hipComplex operator+(const hipComplex& a, const hipComplex& b)
{
	hipComplex r;
	r.x = a.x + b.x;
	r.y = a.y + b.y;
	return r;
}


__global__ void deconvolve(hipComplex *pn, hipComplex *data,
	float *product, int pn_len)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int memIdx;

	if(i >= pn_len) return;

	// move data to local memory
	// Shared mem size = 48kB
	//__shared__ hipComplex smem_data[2080*2];
	__shared__ hipComplex smem_pn[2080];

	// Copy in contiguous chunks of data into SMEM. __sync after each chunk to ensure coalesced access
	for(int memRow = 0; memRow < gridDim.x; memRow++){
		memIdx = memRow * blockDim.x + threadIdx.x;
		if(memIdx < pn_len){
			//smem_data[memIdx] = data[memIdx];
			smem_pn[memIdx] = pn[memIdx];
		}
		__syncthreads();
	}
	// Must sync to ensure all data copied in

	hipComplex s;
	s.x = 0.0;
	s.y = 0.0;
	// Perform deconvolutoin
	for(int n = 0; n < pn_len; n++){
		s = s + (data[n+i] * smem_pn[n]);
	}
	
	// output mag result
	product[i] = magnitude2(s);
}

extern "C"
{	
	hipComplex *d_prncode;
	hipComplex *d_dataold;
	hipComplex *d_dataa;
	hipComplex *d_datab;
	hipComplex *d_datac;
	hipComplex *d_datac_padded;
	hipComplex *d_datad;
	hipfftHandle plan1;
	hipfftHandle plan2;
	float *d_product;
	size_t h_len; ///< length of data in samples


	void start_deconvolve(const complex<float> *h_data, complex<float> *h_product)
	{
		unsigned small_bin_start;
		unsigned small_bin_width = 1040;
		unsigned small_bin_width_padded = small_bin_width * IFFT_PADDING_FACTOR;


		// copy new memory to old
		hipMemcpy(d_dataold, d_dataa, h_len * sizeof(hipComplex), hipMemcpyDeviceToDevice);

		// copy new host data into device memory
		hipMemcpy(d_dataa, h_data, h_len * sizeof(hipComplex), hipMemcpyHostToDevice);

		// perform FFT on spectrum
		hipfftExecC2C(plan1, (hipfftComplex*)d_dataa, (hipfftComplex*)d_datab, HIPFFT_FORWARD);
		hipDeviceSynchronize();

		// shift zero-frequency component to center of spectrum
		small_bin_start = ((16059 + 32768) % 65536);
		// chop spectrum up into 50 spreading channels
		//hipMemcpy(d_datac, d_datab + small_bin_start, 1040 * sizeof(hipComplex), hipMemcpyDeviceToDevice);
		// >> cpy into longer, padded vector
		hipMemcpy(d_datac_padded, d_datab + small_bin_start, 1040 * sizeof(hipComplex), hipMemcpyDeviceToDevice);

		// put back into time domain
		hipfftExecC2C(plan2, (hipfftComplex*)d_datac_padded, (hipfftComplex*)d_datad, HIPFFT_BACKWARD);
		hipDeviceSynchronize();
  		checkCudaErrors(hipGetLastError());
		
	    // Copy [IFFT] results to host
		//hipMemcpy(h_product, d_datad, small_bin_width_padded * sizeof(complex<float>), hipMemcpyDeviceToHost);

		// Task the SM's
		// 1040 * 2 = 2080 samples
		// -> 128 Th/Bl
		// -> ~17 Bl
		deconvolve<<<17, 128>>>(d_prncode, d_dataold, d_product, small_bin_width_padded);
		
		// Copy [deconvolved] results to host
		hipMemcpy(h_product, d_product, small_bin_width_padded * sizeof(float), hipMemcpyDeviceToHost);
	}


	void init_deconvolve(complex<float> *h_pn, size_t len)
	{
		h_len = len;

		// verify that we're a multiple of samples of a thread block
		//if( 0 != (h_len % MAX_THREADS_PER_BLOCK) )
		//	throw runtime_error("[POPGPU] - sample length needs to be multiple of block size.\r\n");

		// allocate CUDA memory
		checkCudaErrors(hipMalloc(&d_prncode, h_len * sizeof(hipComplex)));
		checkCudaErrors(hipMalloc(&d_dataold, h_len * sizeof(hipComplex) * 2));
		d_dataa = d_dataold + h_len; ///< make this sequential to old data
		checkCudaErrors(hipMalloc(&d_product, h_len * sizeof(float)));
		checkCudaErrors(hipMalloc(&d_datab, 655536 * sizeof(hipComplex)));
		checkCudaErrors(hipMalloc(&d_datac, 1040 * sizeof(hipComplex)));
		checkCudaErrors(hipMalloc(&d_datac_padded, 1040 * IFFT_PADDING_FACTOR * sizeof(hipComplex)));
		checkCudaErrors(hipMalloc(&d_datad, 1040 * IFFT_PADDING_FACTOR * sizeof(hipComplex)));

		// initialize CUDA memory
		checkCudaErrors(hipMemcpy(d_prncode, h_pn, h_len, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemset(d_dataold, 0, h_len * sizeof(hipComplex)));
		checkCudaErrors(hipMemset(d_dataa, 0, h_len * sizeof(hipComplex)));		
		checkCudaErrors(hipMemset(d_product, 0, h_len * sizeof(float)));
		checkCudaErrors(hipMemset(d_datac_padded, 0, 1040 * IFFT_PADDING_FACTOR * sizeof(hipComplex)));

	    // setup FFT plans
	    hipfftPlan1d(&plan1, 65536, HIPFFT_C2C, 1);
	    hipfftPlan1d(&plan2, 1040 * IFFT_PADDING_FACTOR, HIPFFT_C2C, 1);
	    printf("[Popwi::popprotadespread]: init deconvolve complete \n");
	}


	//Free all the memory that we allocated
	//TODO: check that this is comprehensive
	void cleanup() {
	  hipfftDestroy(plan1);
	  hipfftDestroy(plan2);
	  checkCudaErrors(hipFree(d_prncode));
	  checkCudaErrors(hipFree(d_dataold));
	  checkCudaErrors(hipFree(d_dataa));
	  checkCudaErrors(hipFree(d_datab));
	  checkCudaErrors(hipFree(d_datac));
	  checkCudaErrors(hipFree(d_datad));
	  checkCudaErrors(hipFree(d_product));
	}

}