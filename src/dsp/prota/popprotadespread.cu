#include "hip/hip_runtime.h"
/******************************************************************************
* Copyright 2013 PopWi Technology Group, Inc. (PTG)
*
* This file is proprietary and exclusively owned by PTG or its associates.
* This document is protected by international and domestic patents where
* applicable. All rights reserved.
*
******************************************************************************/

//#include <hip/hip_complex.h>
#include <complex>
#include <iostream>
#include <stdexcept>
#include "dsp/utils.hpp"

#include <hipfft/hipfft.h>

using namespace std;

#define MAX_THREADS_PER_BLOCK 1024


__device__ float magnitude2( hipComplex& in )
{
	return in.x * in.x + in.y * in.y;
}

__device__ hipComplex operator*(const hipComplex& a, const hipComplex& b)
{
	hipComplex r;
	r.x = b.x*a.x - b.y*a.y;
	r.y = b.y*a.x + b.x*a.y;
	return r;
}

__device__ hipComplex operator+=(const hipComplex& a, const hipComplex& b)
{
	hipComplex r;
	r.x = a.x + b.x;
	r.y = a.y + b.y;
	return r;
}


__global__ void deconvolve(hipComplex *pn, hipComplex *data,
	float *product, int pn_len)
{
	int n;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int temp;

	// move data to local memory
	// Moving in chunks of 1024 samples(4kB). SM = 48kB
	__shared__ hipComplex smem_data[1024];
	__shared__ hipComplex smem_pn[1024]; //TODO: iterate over blocks

	smem_data[threadIdx.x] = data[threadIdx.x];
	smem_pn[threadIdx.x] = pn[threadIdx.x];

	// Must sync to ensure all data copied in
	__syncthreads();

	//hipComplex s = hipComplex(0.0, 0.0);
	hipComplex s;
	s.x = 0.0;
	s.y = 0.0;

	// Perform deconvolutoin
	for( n = 0; n < pn_len; n++)
		temp = n % 1024;
		//s += shrd_data[n + i] * shrd_pn[n];
		s += smem_data[temp] * smem_pn[temp]; // Indexing all wrong here. Computation speed test only


	product[i] = magnitude2(s);
}

extern "C"
{	
	hipComplex *d_prncode;
	hipComplex *d_dataold;
	hipComplex *d_dataa;
	hipComplex *d_datab;
	hipComplex *d_datac;
	hipComplex *d_datad;
	hipfftHandle plan1;
	hipfftHandle plan2;
	float *d_product;
	size_t h_len; ///< length of data in samples


	void start_deconvolve(const complex<float> *h_data, complex<float> *h_product)
	{
		unsigned small_bin_start;
		unsigned small_bin_width = 1040;

		// copy new memory to old
		hipMemcpy(d_dataold, d_dataa, h_len * sizeof(hipComplex), hipMemcpyDeviceToDevice);

		// copy new host data into device memory
		hipMemcpy(d_dataa, h_data, h_len * sizeof(hipComplex), hipMemcpyHostToDevice);

		// Task the SM's
		//deconvolve<<<64, 1024>>>(d_prncode, d_dataold, d_product, h_len);

		// perform FFT on spectrum
		hipfftExecC2C(plan1, (hipfftComplex*)d_dataa, (hipfftComplex*)d_datab, HIPFFT_FORWARD);
		hipDeviceSynchronize();

		// shift zero-frequency component to center of spectrum
		small_bin_start = ((16059 + 32768) % 65536);
		// chop spectrum up into 50 spreading channels
		hipMemcpy(d_datac, d_datab + small_bin_start, 1040 * sizeof(hipComplex), hipMemcpyDeviceToDevice);

		// put back into time domain
		hipfftExecC2C(plan2, (hipfftComplex*)d_datac, (hipfftComplex*)d_datad, HIPFFT_BACKWARD);
		hipDeviceSynchronize();
  		checkCudaErrors(hipGetLastError());
		
	    // Copy results to host
		hipMemcpy(h_product, d_datad, small_bin_width * sizeof(complex<float>), hipMemcpyDeviceToHost);
		//memcpy(h_product, h_data, 1040 * sizeof(complex<float>));
	}


	void init_deconvolve(complex<float> *h_pn, size_t len)
	{
		h_len = len;

		// verify that we're a multiple of samples of a thread block
		if( 0 != (h_len % MAX_THREADS_PER_BLOCK) )
			throw runtime_error("[POPGPU] - sample length needs to be multiple of block size.\r\n");

		// allocate CUDA memory
		checkCudaErrors(hipMalloc(&d_prncode, h_len * sizeof(hipComplex)));
		checkCudaErrors(hipMalloc(&d_dataold, h_len * sizeof(hipComplex) * 2));
		d_dataa = d_dataold + h_len; ///< make this sequential to old data
		checkCudaErrors(hipMalloc(&d_product, h_len * sizeof(float)));
		checkCudaErrors(hipMalloc(&d_datab, 655536 * sizeof(hipComplex)));
		checkCudaErrors(hipMalloc(&d_datac, 1040 * sizeof(hipComplex)));
		checkCudaErrors(hipMalloc(&d_datad, 1040 * sizeof(hipComplex)));

		// initialize CUDA memory
		checkCudaErrors(hipMemcpy(d_prncode, h_pn, h_len, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemset(d_dataold, 0, h_len * sizeof(hipComplex)));
		checkCudaErrors(hipMemset(d_dataa, 0, h_len * sizeof(hipComplex)));		
		checkCudaErrors(hipMemset(d_product, 0, h_len * sizeof(float)));

	    // setup FFT plans
	    hipfftPlan1d(&plan1, 65536, HIPFFT_C2C, 1);
	    hipfftPlan1d(&plan2, 1040, HIPFFT_C2C, 1);
	}

	//Free all the memory that we allocated
	//TODO: check that this is comprehensive
	void cleanup() {
	  hipfftDestroy(plan1);
	  hipfftDestroy(plan2);
	  checkCudaErrors(hipFree(d_prncode));
	  checkCudaErrors(hipFree(d_dataold));
	  checkCudaErrors(hipFree(d_dataa));
	  checkCudaErrors(hipFree(d_datab));
	  checkCudaErrors(hipFree(d_datac));
	  checkCudaErrors(hipFree(d_datad));
	  checkCudaErrors(hipFree(d_product));
	}

}