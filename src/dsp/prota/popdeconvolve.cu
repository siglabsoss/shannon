#include "hip/hip_runtime.h"
/******************************************************************************
* Copyright 2013 PopWi Technology Group, Inc. (PTG)
*
* This file is proprietary and exclusively owned by PTG or its associates.
* This document is protected by international and domestic patents where
* applicable. All rights reserved.
*
******************************************************************************/

#ifndef __POP_DECONVOLVE_CU__
#define __POP_DECONVOLVE_CU__

//#include <popComplex.h>
#include <complex>
#include <iostream>
#include <stdexcept>
#include <cstdio>
#include <cmath>
#include "dsp/utils.hpp"

#include <hipfft/hipfft.h>
#include <cstdlib>
#include <time.h>
//#include "cuPrintf.cu"
//#include "shrUtils.h"
//#include "cutil_inline.h"
#include <dsp/common/poptypes.cuh>

#include <dsp/prota/popchanfilter.cuh>
#include <dsp/prota/popdeconvolve.cuh>

using namespace std;


__global__ void threshold_detection(popComplex *in, int *out, unsigned int *outLen, int outLenMax, double thresholdSquared, int len, int fbins)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

//	int I = blockDim.x * gridDim.x;
//
//	int F = (I / len);
//	int f = (i / len) - (F / 2); // frequency
	int b = i % len; // fft bin

	double mag; // magnitude of peak

	// don't look for peaks in padding
	if( (b > (len / 4)) && (b <= (3 * len /4)) ) return;

	// take the magnitude of the detection
	mag = magnitude2(in[i]);

	// if the magnitude is below the thresh, return
	if( mag < thresholdSquared ) return;

	// atomicInc increments the variable at the pointer only if the second param is larger than the stored variable
	// this variable always starts at 0 (set before the kernel launch)
	// the "old" value at the pointer location is returned, which is this thread's unique index into the output buffer
	int ourUniqueIndex = atomicInc(outLen, INT_MAX);

	// out of bounds, this is an OVERFLOW ie we found too many peaks
	// in this case we discard the data, but outLen is still incremented.  we can check for overflow after the kernel launch is done
	if( ourUniqueIndex > outLenMax ) return;

	// save the index of our detection to the array
	out[ourUniqueIndex] = i;
}

#define CHECK_POINTS (8)

// data is raw complex double float samples
// in is an array of detected peaks
// inLen is the count of detected peaks
// out is an array of detected local maxima pkeas
// outLen is the number of detected local maxima peaks

__global__ void local_maxima_detection(popComplex *data, int *in, unsigned int *inLen, int *out, unsigned int *outLen, popComplex* d_maxima_peaks_neighbors, unsigned peak_sinc_neighbors, int outLenMax, int spreadLength, int fbins)
{
	// detectedPeakIndex is the index into in[] which this thread is looking at
	int detectedPeakIndex = blockIdx.x * blockDim.x + threadIdx.x;

	// don't process if this thread is looking at a peak that wasn't detected
	if( detectedPeakIndex > *inLen )
		return;

	// sample is the index into data which we are considering for a local maxima
	int sampleIndex = in[detectedPeakIndex];

	double surroundingMax = 0.0; // maximum of surrounding points

	int check[CHECK_POINTS];

	check[0] = sampleIndex + 1; // right
	check[1] = sampleIndex - 1; // left
	check[2] = sampleIndex - spreadLength; // up
	check[3] = sampleIndex + spreadLength; // down
	check[4] = sampleIndex - spreadLength - 1; // up left
	check[5] = sampleIndex - spreadLength + 1; // up right
	check[6] = sampleIndex + spreadLength - 1; // down left
	check[7] = sampleIndex + spreadLength + 1; // down right

	int testSample;

	for( int i = 0; i < CHECK_POINTS; i++ )
	{
		// bound to edges of array
		testSample = min(max(check[i],0), spreadLength*fbins);

		// compute maximum of surrounding indices
		surroundingMax = fmax(surroundingMax, magnitude2(data[testSample]));
	}


	// bail if not a local maxima
	if ( magnitude2(data[sampleIndex]) <= surroundingMax )
		return;


	// atomicInc increments the variable at the pointer only if the second param is larger than the stored variable
	// this variable always starts at 0 (set before the kernel launch)
	// the "old" value at the pointer location is returned, which is this thread's unique index into the output buffer
	int ourUniqueIndex = atomicInc(outLen, INT_MAX);

	// out of bounds, this is an OVERFLOW ie we found too many peaks
	// in this case we discard the data, but outLen is still incremented.  we can check for overflow after the kernel launch is done
	if( ourUniqueIndex > outLenMax ) return;

	// save the index of our detection to the array
	out[ourUniqueIndex] = sampleIndex;

	// copy neighbors into this array;
	// we copy peak_sinc_neighbors (8) samples on either side for a total of (17) samples
	// after this kernel is done the host can do a single memcopy and get all the needed information for sinc interpolation
	memcpy(d_maxima_peaks_neighbors, data+sampleIndex-peak_sinc_neighbors, (1+peak_sinc_neighbors+peak_sinc_neighbors)*sizeof(popComplex) );
}

#undef CHECK_POINTS


extern "C"
{	

// d_out is an array of samples which are above the threshold with size outLenMax
// d_outLen is the length of valid samples in the d_out array (with a value of no more than outLenMax)
// d_maxima_out is an array of samples which have passed the local maxima test
	void gpu_threshold_detection(popComplex* d_in, int* d_out, unsigned int *d_outLen, int* d_maxima_out, unsigned int *d_maxima_outLen, popComplex* d_maxima_peaks_neighbors, unsigned peak_sinc_neighbors, int outLenMax, double threshold, int len, int fbins, hipStream_t* stream)
	{
		// reset this index of the largest detected peak to 0
		checkCudaErrors(hipMemsetAsync(d_outLen, 0, sizeof(int), *stream));

		threshold_detection<<<fbins * 16, len / 16, 0, *stream>>>(d_in, d_out, d_outLen, outLenMax, (threshold*threshold), len, fbins);

		checkCudaErrors(hipMemsetAsync(d_maxima_outLen, 0, sizeof(int), *stream));

		local_maxima_detection<<<1, outLenMax, 0, *stream>>>(d_in, d_out, d_outLen, d_maxima_out, d_maxima_outLen, d_maxima_peaks_neighbors, peak_sinc_neighbors, outLenMax, len, fbins);

	}




//	void gpu_peak_detection(popComplex* in, double* peak, int len, int fbins)
//	{
//		// TODO: better refactor thread and block sizes for any possible spreading code and fbin lengths
//		peak_detection<<<fbins * 16, len / 16>>>(in, peak, len);
//		hipDeviceSynchronize();
//	}


}

#endif
